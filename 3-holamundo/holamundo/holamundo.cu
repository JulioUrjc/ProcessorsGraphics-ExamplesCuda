#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 500000
#define BLOCK_WIDTH 500

__global__ void kernel(char * m){

	printf("%s soy el thread %d \n",m, threadIdx.x);
   return;
}

int main (int argc, char ** argv) {


   int tamM = 8;
   char * m  = (char *) malloc(tamM);
   char * dm = NULL;

   _snprintf_s(m,8,8, "Hello!!\0");

   fprintf(stderr,"Al hacer la reserva: %s \n",hipGetErrorString(hipMalloc(&dm,tamM)));
   fprintf(stderr,"Al hacer la copia:   %s \n",hipGetErrorString(hipMemcpy(dm,m,tamM,hipMemcpyHostToDevice)));
   fprintf(stderr,"Soy el HOST voy a mandar a dispositivo: %s\n",m);

   //Launch kernel
   kernel<<< 1, 16 >>>(dm);

   int dev_count;
   hipDeviceProp_t prop;
   hipGetDeviceCount(&dev_count);
   for (int i = 0; i < dev_count; i++){
	   hipGetDeviceProperties(&prop, i);
	   //fprintf(stderr, " %s\n", prop);
   }

   fprintf(stderr,"Mensaje: %s\n",m);

   hipFree(dm);
   
   return 0;
}
