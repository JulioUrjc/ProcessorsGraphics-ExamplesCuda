#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(char * m){


   return;
}

int main (int argc, char ** argv) {


   int tamM = 8;
   char * m  = (char *) malloc(tamM);
   char * dm = NULL;

   _snprintf_s(m,8,8, "Hello!!\0");

   fprintf(stderr,"Al hacer la reserva: %s \n",hipGetErrorString(hipMalloc(&dm,tamM)));
   fprintf(stderr,"Al hacer la copia:   %s \n",hipGetErrorString(hipMemcpy(dm,m,tamM,hipMemcpyHostToDevice)));
   fprintf(stderr,"Soy el HOST voy a mandar a dispositivo: %s\n",m);

   //Launch kernel


   fprintf(stderr,"Mensaje: %s\n",m);

   hipFree(dm);
   
   return 0;
}
